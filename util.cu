#include "hip/hip_runtime.h"
#include "util.cuh"
#include "nbody.cuh"

#include <string>

float prevX = WINDOW_W/2, prevY = WINDOW_H/2;
bool mouseUp = 0;
bool toggleHelp = true;

extern float3 pos[N_SIZE];
extern float3 vel[N_SIZE];
extern float3 acc[N_SIZE];
extern float m[N_SIZE];
extern float r[N_SIZE];

GLfloat lpos[4] = {-0.3,0.0,200,0}; //Positioned light
GLfloat light_specular[4] = {1, 0.6, 1, 0}; //specular light intensity (color)
GLfloat light_diffuse[] = { 1.0, 1.0, 1.0, 0.0 };//diffuse light intensity (color)
GLfloat light_ambient[] = { 0.2, 0.2, 0.2, 0.0 }; //ambient light intensity (color)
GLfloat a;
GLfloat mat_emission[] = {0.8, 0.5, 0.3, 0.0}; //object material preperty emission of light
GLfloat mat_specular[] = { 4.0, 0.5, 2.0, 0.0 }; //object material specularity
GLfloat low_shininess[] = { 50 };
GLfloat fogColor[] = {0.5f, 0.5f, 0.5f, 1};

void timerFunc(int value)
{
    glutPostRedisplay();
}

void resizeCallback(int w, int h) {
    if( ORTHO_VERSION ) return;
    // Prevent a divide by zero, when window is too short
    // (you cant make a window of zero width).
    if(h == 0)
        h = 1;

    float ratio = 1.0* w / h;

    // Reset the coordinate system before modifying
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    
    // Set the viewport to be the entire window
    glViewport(0, 0, w, h);

    // Set the correct perspective.
    gluPerspective(45,ratio,1,1000);
    glMatrixMode(GL_MODELVIEW);
    
}


void keyboardFunc(unsigned char key, int x, int y) {

    if (key == 27) 
        exit(0);

    float vel = 5.0;
    float rightX, rightY, rightZ;
    cross(camera.forwardX,camera.forwardY,camera.forwardZ, camera.upX,camera.upY, camera.upZ, rightX, rightY, rightZ);
    float sizeRight = sqrtf(rightX*rightX + rightY*rightY + rightZ*rightZ);
    rightX /= sizeRight; rightY /= sizeRight; rightZ /= sizeRight;

    if( key == 'w' ) // move forward
    {
        camera.camX += camera.forwardX*vel;
        camera.camY += camera.forwardY*vel;
        camera.camZ += camera.forwardZ*vel;
    }
    if( key == 's' ) // move backward
    {
        camera.camX -= camera.forwardX*vel;
        camera.camY -= camera.forwardY*vel;
        camera.camZ -= camera.forwardZ*vel;
    }
    if( key == 'a' ) // move left
    {

        camera.camX -= rightX*vel;
        camera.camY -= rightY*vel;
        camera.camZ -= rightZ*vel;
    }
    if( key == 'd' ) // move right
    {
        camera.camX += rightX*vel;
        camera.camY += rightY*vel;
        camera.camZ += rightZ*vel;
    }

    if( key == 'h' ) // show or hide help
    {
        toggleHelp = !toggleHelp;
    }


}
void PassiveMouseMotion( int x, int y ){
    prevX = x, prevY = y;

}

// call back function triggered by mouse
void mouseCallback(int x, int y){

        float velx = (float(x -prevX)/WINDOW_W);
        float vely = (float(y -prevY)/WINDOW_H);
        prevX = x;
        prevY = y;
        camera.phi += -velx*M_PI*0.9;
        camera.theta += -vely*M_PI*0.9;

        float rightX, rightY, rightZ;
        rightX = sinf(camera.phi - M_PI/2.0f);
        rightY = 0;
        rightZ = cosf(camera.phi - M_PI/2.0f);
        float sizeRight = sqrtf(rightX*rightX + rightY*rightY + rightZ*rightZ);
        rightX /= sizeRight; rightY /= sizeRight; rightZ /= sizeRight;


        camera.forwardX = cosf(camera.theta)*sinf(camera.phi);
        camera.forwardY = sinf(camera.theta);
        camera.forwardZ = cosf(camera.theta)*cosf(camera.phi);

        float sizeForward = sqrtf(camera.forwardX*camera.forwardX + camera.forwardY*camera.forwardY + camera.forwardZ*camera.forwardZ);
        camera.forwardX /= sizeForward; camera.forwardY /= sizeForward; camera.forwardZ /= sizeForward;

        float newUpX, newUpY, newUpZ;

        cross(rightX,rightY,rightZ, camera.forwardX,camera.forwardY, camera.forwardZ, newUpX, newUpY, newUpZ);
        float sizeUp = sqrtf(newUpX*newUpX + newUpY*newUpY + newUpZ*newUpZ);
        camera.upX = newUpX/sizeUp; camera.upY = newUpY/sizeUp; camera.upZ = newUpZ/sizeUp;
        


}

void cross(float x1, float y1, float z1, float x2, float y2, float z2,float& rightX, float& rightY, float& rightZ){
    rightX = y1*z2 - z1*y2;
    rightY = x1*z2 - x1*z2;
    rightZ = x1*y2 - y1*x1;

}

 

void DrawCircle(float cx, float cy, float r, int num_segments) {
    glBegin(GL_LINE_LOOP);
    for (int ii = 0; ii < num_segments; ii++)   {
        float theta = 2.0f * PI * float(ii) / float(num_segments);//get the current angle 
        float x = r * cosf(theta);//calculate the x component 
        float y = r * sinf(theta);//calculate the y component 
        glVertex2f(x + cx, y + cy);//output vertex 
    }
    glEnd();
}

void drawText(std::string text, float x, float y){
    glMatrixMode(GL_MODELVIEW);
    glPushMatrix();


    glColor3f(1.0f, 0.0f, 0.0f);//needs to be called before RasterPos
    glRasterPos2f(x, y);

    
    void * font = GLUT_BITMAP_TIMES_ROMAN_24;

    for (std::string::iterator i = text.begin(); i != text.end(); ++i)
    {
        char c = *i;
        glutBitmapCharacter(font, c);
    }
    glPopMatrix();
}


void setLights(){


    glMaterialfv(GL_FRONT, GL_EMISSION, mat_emission);
    glMaterialfv(GL_FRONT, GL_SPECULAR, mat_specular);
    glMaterialfv(GL_FRONT, GL_SHININESS, low_shininess);
    glLightfv(GL_LIGHT0, GL_SPECULAR, light_specular);
    glLightfv(GL_LIGHT0, GL_POSITION, lpos);


    //Adding fog
    glFogfv(GL_FOG_COLOR, fogColor);
    glFogi(GL_FOG_MODE, GL_LINEAR);
    glFogf(GL_FOG_START, 10.0f);
    glFogf(GL_FOG_END, 1000.0f);
}

void draw2(){
    glClearColor(0.1f,0.1f,0.1f,0.1f);
    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
    

    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
    if( !ORTHO_VERSION ) {
        gluLookAt(camera.camX,camera.camY,camera.camZ, //Camera position
              camera.camX+camera.forwardX,camera.camY+camera.forwardY,camera.camZ+camera.forwardZ, //Position of the object to look at
              camera.upX,camera.upY,camera.upZ); //Camera up direction
    }
    setLights();

    runKernelNBodySimulation();

    if( toggleHelp ){
        if( !ORTHO_VERSION )
        {
            drawText("USAGE INFO", 50,60);
            drawText("Use keys w, a, s, d to move", 50,50);
            drawText("Hold the left button on the mouse to look around", 50,40);
            drawText("Press h to show/hide this help info", 50,30);
        }
        else
        {
            drawText("USAGE INFO", 50,80);
            drawText("Use keys W, A, S, D to move", 50,60);
            drawText("Hold the left button on the mouse to look around", 50,40);
            drawText("Press H to show/hide this help info", 50,10);
        }
    }
    

    glColor3f(0.5f, 0.5f, 0.3f);
    for(int i = 0; i < N_SIZE; i ++){
        if(m[i]>0)
        {
            if( !ORTHO_VERSION ) 
            {
                glPushMatrix();
                glTranslatef(pos[i].x, pos[i].y,pos[i].z);
                glutSolidSphere(r[i],10,10); // draw sphere
                glPopMatrix();
            }
            else{
                DrawCircle(pos[i].x, pos[i].y, r[i], 10); // draw circle
            }
        }
            
    }

    glutSwapBuffers();


}





 
