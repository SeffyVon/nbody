#include "hip/hip_runtime.h"
// This is the REAL "hello world" for CUDA!
// It takes the string "Hello ", prints it, then passes it to CUDA with an array
// of offsets. Then the offsets are added in parallel to produce the string "World!"
// By Ingemar Ragnemalm 2010
 
#include <stdio.h>
#include <GL/glut.h>
 
const int N = 16; 
const int blocksize = 16; 
 
__global__ 
void hello(char *a, int *b) 
{
	a[threadIdx.x] += b[threadIdx.x];
}
 
int mainTestCUDA()
{
	char a[N] = "Hello \0\0\0\0\0\0";
	int b[N] = {15, 10, 6, 0, -11, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
 
	char *ad;
	int *bd;
	const int csize = N*sizeof(char);
	const int isize = N*sizeof(int);
 
	printf("%s", a);
 
	hipMalloc( (void**)&ad, csize ); 
	hipMalloc( (void**)&bd, isize ); 
	hipMemcpy( ad, a, csize, hipMemcpyHostToDevice ); 
	hipMemcpy( bd, b, isize, hipMemcpyHostToDevice ); 
	
	dim3 dimBlock( blocksize, 1 );
	dim3 dimGrid( 1, 1 );
	hello<<<dimGrid, dimBlock>>>(ad, bd);
	hipMemcpy( a, ad, csize, hipMemcpyDeviceToHost ); 
	hipFree( ad );
	hipFree( bd );
	
	printf("%s\n", a);
	return EXIT_SUCCESS;
}



void draw(void) {

    // Black background
    glClearColor(0.0f,0.0f,0.0f,1.0f);
    glClear(GL_COLOR_BUFFER_BIT);
    //Draw i
    glFlush();

    mainTestCUDA();

}

//Main program
int main(int argc, char **argv) {

    glutInit(&argc, argv);

    /*Setting up  The Display
    /    -RGB color model + Alpha Channel = GLUT_RGBA
    */
    glutInitDisplayMode(GLUT_RGBA|GLUT_SINGLE);

    //Configure Window Postion
    glutInitWindowPosition(50, 25);

    //Configure Window Size
    glutInitWindowSize(480,480);

    //Create Window
    glutCreateWindow("Hello OpenGL");


    //Call to the drawing function
    glutDisplayFunc(draw);

    // Loop require by OpenGL
    glutMainLoop();
    return 0;
}
