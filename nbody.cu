#include "hip/hip_runtime.h"

#include "nbody.cuh"

int bodies_size = 0;
Body *bodies_dev = NULL;
Body bodies[N_SIZE] =  {  Body(1,0,4,4), Body(13,2,0,1),  Body(8,3,4,2), Body(2,7,3,2), Body(1,2,0,4), Body(3,2,0,3),  Body(4,3,4,2), Body(2,4,3,2) };


void initCUDA()
{
	bodies_size = N_SIZE * sizeof(Body);
	hipMalloc( (void**)&bodies_dev, bodies_size ); 
	hipMemcpy( bodies_dev, bodies, bodies_size, hipMemcpyHostToDevice );
}

void init()
{
	initCUDA();
	atexit(deinit);
}

void deinit()
{
	hipFree( bodies_dev );
}

__device__ 
void updateAcceleration(Body &body){
	body.a.x ++;
	body.a.y ++;
	body.a.z ++;
}

__device__
void updateVelocity(Body &body){
	body.v.x += body.a.x;
	body.v.y += body.a.y;
	body.v.z += body.a.z;
}

__device__
void updatePosition(Body &body){
	body.pos.x += body.v.x;
	body.pos.y += body.v.y;
	body.pos.z += body.v.z;
}
 
__global__ 
void nbody(Body *body) 
{
	updateAcceleration(body[threadIdx.x]);
	updateVelocity(body[threadIdx.x]);
	updatePosition(body[threadIdx.x]);
}

int runKernelNBodySimulation()
{

	nbody<<<GRID_SIZE, BLOCK_SIZE>>>(bodies_dev);
	hipMemcpy( bodies, bodies_dev, bodies_size, hipMemcpyDeviceToHost ); 

	for(int i = 0; i < N_SIZE; i++){
		printf("a=(%f,%f,%f)\n", bodies[i].a.x, bodies[i].a.y, bodies[i].a.z);
	}
	

	return EXIT_SUCCESS;
}